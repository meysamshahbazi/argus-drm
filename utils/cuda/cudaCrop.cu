#include "hip/hip_runtime.h"
#include "cudaCrop.h"


__global__ void DownsampleGPU(uchar4* srcImage, uchar4* dstImage, size_t src_width, size_t src_height, size_t src_pitch,
	int x0, int y0, int df, size_t dst_width, size_t dst_height)
{
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	if( x >= dst_width || y >= dst_height  )
		return; 

	const int pixel_src = (y*df+y0) * src_pitch + (x*df+x0);
	const int pixel_dst = y * dst_width + x;

	dstImage[pixel_dst] = srcImage[pixel_src];
}



hipError_t cudaDownsample( uchar4* srcDev, uchar4* dstDev, size_t src_width, size_t src_height,size_t src_pitch,
	int x0, int y0, int df, size_t dst_width, size_t dst_height, hipStream_t stream)
{
	if( !srcDev || !dstDev )
		return hipErrorInvalidDevicePointer;

	if( src_width == 0 || src_height == 0)
		return hipErrorInvalidValue;

	const dim3 blockDim(32,8,1);
	const dim3 gridDim(iDivUp(dst_width,blockDim.x), iDivUp(dst_height,blockDim.y), 1);

	DownsampleGPU<<<gridDim, blockDim,0,stream>>>( srcDev, dstDev, src_width, src_height, src_pitch, x0, y0, df, dst_width, dst_height);
	
	return CUDA(hipGetLastError());
}